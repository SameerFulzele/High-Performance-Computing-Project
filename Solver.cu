#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>

__global__ void step1(float* fn2, float* fn1, float B)
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = blockId *(blockDim.x* blockDim.y)+(threadIdx.y *blockDim.x)+threadIdx.x;
	int id1 = threadId + 1;
	int id2 = threadId - 1;
	int id3 =  threadId - blockDim.x ;
	int id4 = threadId + blockDim.x ;
	if(threadIdx.y == 0)
	{
		int blockIdof3 = blockIdx.x + (blockIdx.y -1) * gridDim.x;
		id3 = blockIdof3 *(blockDim.x* blockDim.y)+((blockDim.y-1) *blockDim.x)+threadIdx.x;
	}

	if(threadIdx.y == blockDim.y -1 )
	{
		int blockIdof4 = blockIdx.x + (blockIdx.y +1) * gridDim.x;
		id4 = blockIdof4 *(blockDim.x* blockDim.y) + threadIdx.x;
	}


	if(threadIdx.x == 0)
	{
		int blockIdof2 = (blockIdx.x-1) + blockIdx.y * gridDim.x;
		id2 = blockIdof2 *(blockDim.x* blockDim.y)+(threadIdx.y *blockDim.x)+(blockDim.x-1);
	}

	if(threadIdx.x == blockDim.x -1 )
	{
		int blockIdof1 = (blockIdx.x+1) + blockIdx.y * gridDim.x;
		id1 = blockIdof1 *(blockDim.x* blockDim.y)+(threadIdx.y *blockDim.x);
	}

	if(threadIdx.y == 0 && blockIdx.y == 0)
		id3 = threadId;

	if(threadIdx.y == blockDim.y -1  && blockIdx.y == gridDim.y -1)
		id4 = threadId;

	if(threadIdx.x == 0 && blockIdx.x == 0)
		id2 = threadId;

	if(threadIdx.x == blockDim.x -1  && blockIdx.x == gridDim.x -1)
		id1 = threadId;

	fn2[threadId] = 2*fn1[id]+B*(fn1[id1]+ fn1[id2]+ fn1[id3] + fn1[id4] - (4*fn1[id]) );
}

__global__ void step2(float* fn2, float* fn0)
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = blockId *(blockDim.x* blockDim.y)+(threadIdx.y *blockDim.x)+threadIdx.x;

	fn2[blockId ] = fn2[blockId] - fn0[blockId] ;
}
__global__ void copy(float* fn1, float* fn0)
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = blockId *(blockDim.x* blockDim.y)+(threadIdx.y *blockDim.x)+threadIdx.x;

	fn1[blockId] = fn0[blockId];
}

int main()
{

	int N = 1028;
	int T = 16;
	int G = 64;
	int size = N*N* sizeof(int);

	float f0[N*N], f1[N*N], f2[N*N],*fn1, *fn2,*fn0;
	float h = 0.001, At = 0.1, c = 0.01, b;
	b = (c*c*At*At) / (h*h);
	b= 0.0025;

	int i,j;
	for(i=0; i<16; i++ )
	{
		for(j=0; j<16; j++ )
		{
			f0[j + (i*N)] = 0;
			f1[j + (i*N)] = 0;
			f2[j + (i*N)] = 0;
		}
	}
	int x = N*N/2;
	f1[x] = 5;

	hipMalloc(&fn1, size);
	hipMemcpy(fn1, f1 , size, hipMemcpyHostToDevice);

	hipMalloc(&fn0, size);
	hipMemcpy(fn0, f0, size, hipMemcpyHostToDevice);
	hipMalloc(&fn2, size);

	dim3   DimBlock(T,T);
	dim3   DimGrid(G,G);

	int n;      // n is the number of iteration
	n = 4;

	for(i=0 ;i<n ; i++)
	{
		step1<<< DimGrid,DimBlock >>>(fn2 , fn1 ,b);
		step2<<<DimGrid,DimBlock >>>(fn2 , fn1);

		copy<<< DimGrid,DimBlock >>>(fn0 , fn1 );
		copy<<< DimGrid,DimBlock >>>(fn1 , fn2 );
	}

	hipMemcpy(f2, fn2, size, hipMemcpyDeviceToHost);
	hipMemcpy(f1, fn1, size, hipMemcpyDeviceToHost);
	hipMemcpy(f0, fn0, size, hipMemcpyDeviceToHost);
	indexof_f2 = ( ((j/T) + (i/T) * T) * (G*G) ) + j%T + ((i%t)*G);

	for(i=0 ; i<N ; i++)
		{
			for(j=0;j<N;j++){
				printf("%f  ",f2[indexof_f2]);
			}
			printf("\n");
		}
}
