
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void step1(float* fn2, float* fn1, float B)
{
	int id = (threadIdx.y * blockDim.x) + threadIdx.x + (blockIdx.x * blockDim.x * blockDim.y  ); // here 256  is  blockDim.x * blockDim.y  
	int id1 = id + 1 ;						
	int id2 = id -1 ;
	
	
int id3 =  (threadIdx.y * blockDim.x) + threadIdx.x - blockDim.x + (blockIdx.x * blockDim.x * blockDim.y  );  
      int id4 =(threadIdx.y * blockDim.x) + threadIdx.x + blockDim.x + (blockIdx.x * blockDim.x * blockDim.y  ) ;
	if(threadIdx.y == 0 && blockIdx.x ==0 )
     { id3 = id ; } 

	if(threadIdx.y == blockDim.y -1 && blockIdx.x == gridDim.x - 1)
     { id4 = id ; } 
	
	
        fn2[id] = 2*fn1[id]+B*(fn1[id1]+ fn1[id2]+ fn1[id3] + fn1[id4] - (4*fn1[id]) );
}






__global__ void step2(float* fn2, float* fn0)
{
	int id= threadIdx.y * blockDim.x + threadIdx.x + blockIdx.x * blockDim.x * blockDim.y  ;
	  

	fn2[id ] = fn2[id]+ fn0[id] ;
}
__global__ void copy(float* fn1, float* fn0)
{
	int id = threadIdx.y * blockDim.x + threadIdx.x +  (blockIdx.x * blockDim.x *blockDim.y) ;

	fn1[id] = fn0[id] ;
} 

					  



int main()
{
	int size = 256*256* sizeof(int);
	//int N = 256*256 ;
	float f0[256][256], f1[256][256], f2[256][256],*fn1, *fn2,*fn0;
	

	float h = 0.001  , At = 0.1 , c = 0.01  ,b ;
	b = (c*c*At*At) / (h*h) ;
	int i,j;
	for(i=0; i<256; i++ )
	{
		for(j=0; j<256; j++ )
		{
			f0[i][j] = 0;
			f1[i][j] = 0;
			f2[i][j] = 0;
		}
	}
/*for(i=120; i<156; i++ )
	{
		for(j=120; j<156; j++ )
		{
			f1[i][j] = 3;
			
		}
	}*/
	f1[125][125] = 5 ;
	hipMalloc(&fn1, size);
	hipMemcpy(fn1, f1 , size, hipMemcpyHostToDevice); // if this doesn't work make f0,f1,f2 a 1D array 

	hipMalloc(&fn0, size);
	hipMemcpy(fn0, f0, size, hipMemcpyHostToDevice);

	hipMalloc(&fn2, size);

	     
	dim3   DimBlock(16,16);   // each block will have 16 * 16 threads so total elements N=256*256 total no of blocks is N/256 = 256
	
	//if we want values of matrix at time t = T sec 
	

	for(i=0 ;i<2 ; i++){
	step1<<< 256,DimBlock >>>(fn2 , fn1 ,b);
	step2<<< 256,DimBlock >>>(fn2 , fn1);
	
		
	copy<<< 256,DimBlock >>>(fn0 , fn1 );
	copy<<< 256,DimBlock >>>(fn1 , fn2 );}
		
		hipMemcpy(f2, fn2, size, hipMemcpyDeviceToHost);
		hipMemcpy(f1, fn1, size, hipMemcpyDeviceToHost);
				hipMemcpy(f0, fn0, size, hipMemcpyDeviceToHost);
    
	for(i=120; i<130; i++ )
	{
		for(j=120; j<130; j++ )
		{
			printf("\t %f",f2[i][j]);
		}
		printf("\n ");
	} 

	printf("\n  %f",b);

	for(i=120; i<130; i++ )
	{
		for(j=120; j<130; j++ )
		{
			printf("\t %f",f1[i][j]);
		}
		printf("\n ");
	} 
		printf("\n  %f",b);
	
	

	for(i=120; i<130; i++ )
	{
		for(j=121; j<128; j++ )
		{
			printf("\t %f",f0[i][j]);
		}
		printf("\n ");
	} 
		printf("\n  %f",b);
	
}
