
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void step1(float* fn2, float* fn1, float B)
{
	int id = (threadIdx.y * blockDim.x) + threadIdx.x ;  
	int id1 = id + 1 ;						
	int id2 = id - 1  ;
	
	
     	 int id3 =  id - blockDim.x ;  
     	 int id4 = id + blockDim.x ;
	if(threadIdx.y == 0)
   	  { id3 = id ; } 

	if(threadIdx.y == blockDim.y -1 )
  	  { id4 = id ; } 
	
	
        fn2[id] = 2*fn1[id]+B*(fn1[id1]+ fn1[id2]+ fn1[id3] + fn1[id4] - (4*fn1[id]) );	

	
}






__global__ void step2(float* fn2, float* fn0)
{
	int id= threadIdx.y * blockDim.x + threadIdx.x + blockIdx.x * blockDim.x * blockDim.y  ;
	  

	fn2[id ] = fn2[id] - fn0[id] ;
}
__global__ void copy(float* fn1, float* fn0)
{
	int id = threadIdx.y * blockDim.x + threadIdx.x +  (blockIdx.x * blockDim.x *blockDim.y) ;

	fn1[id] = fn0[id] ;
} 

					  



int main()
{	

	int N = 16 ;
	int size = N*N* sizeof(int);
	
	float f0[N*N], f1[N*N], f2[N*N],*fn1, *fn2,*fn0;
	

	float h = 0.001  , At = 0.1 , c = 0.01  ,b ;
	b = (c*c*At*At) / (h*h) ;
	b= 0.0025;
	


	
	int i,j;
	for(i=0; i<N; i++ )
	{
		for(j=0; j<N; j++ )
		{
			f0[j + (i*N)] = 0;
			f1[j + (i*N)] = 0;
			f2[j + (i*N)] = 0;
		}

	}
	int x = N*N/2;
	f1[x] = 5 ;






	hipMalloc(&fn1, size);
	hipMemcpy(fn1, f1 , size, hipMemcpyHostToDevice); 

	hipMalloc(&fn0, size);
	hipMemcpy(fn0, f0, size, hipMemcpyHostToDevice);

	hipMalloc(&fn2, size);

	     
	dim3   DimBlock(N,N);   
	
	int n;      // n is the number of iteration 
	n = 4 ;
	


	for(i=0 ;i<n ; i++)
	{
	
	step1<<< 1,DimBlock >>>(fn2 , fn1 ,b);
	step2<<< 1,DimBlock >>>(fn2 , fn1);
	
		
	copy<<< 1,DimBlock >>>(fn0 , fn1 );
	copy<<< 1,DimBlock >>>(fn1 , fn2 );
	
	}
		
		


		hipMemcpy(f2, fn2, size, hipMemcpyDeviceToHost);
		hipMemcpy(f1, fn1, size, hipMemcpyDeviceToHost);
		hipMemcpy(f0, fn0, size, hipMemcpyDeviceToHost);
    
	for(i=N; i<N; i++ )
	{
		for(j=0; j<N; j++)
		{	
			printf("%f ",f2[j + (i*N)]);
			
		}
		printf("\n END ");}

		for(j=N-2; j<N+3; j++)
		{	
			printf("%f   ",f2[j]);
			
		}

	
printf("%f  %f \n inti %f   ",f2[N*7],f2[N*6],f2[x]);
	


	
		
}
