
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

/*
Our equation is 
F(i,j,t+1) = 2*F(i,j,t)  + B*B [F(i+1,j,t) + F(i-1,j,t) + F(i,j+1,t) + F(i,j-1,t) - 4*F(i,j,t) ] + F(i,j,t-1)  

We use step 1 kernel to evaluate first two terms in RHS and then step 2 kernel to add last 3rd term in RHS to F(i,j,t+1) in LHS

Here B =  (C*C) * (deltaT*deltaT) / (h*h) where h is the seperation between two points 
				  	and deltaT is the time period between two iterations





If we use 2D grid and 2D block in we will have indexing like 

blockId = blockIdx.x + blockIdx.y * gridDim.x;
threadId = blockId *(blockDim.x* blockDim.y)+(threadIdx.y *blockDim.x)+threadIdx.x;

this threadId will be F(i,j,t) to find the threadId of F(i,j+1,t), F(i,j-1,t) , F(i-1,j,t) , F(i+1,j,t)  


1st case : if all the 4 are inside the block and not at the boundary of the block 
we have id1, id2, id3, id4 for  F(i,j+1,t), F(i,j-1,t) , F(i-1,j,t) , F(i+1,j,t) respectively 

2nd case : if all four lie on the boundary of the block but block is not on the boundary of the initial matrix
for id1,id2,id3,id4 we need to acess the id of the blocks nearest to them



3rd case : where we have all four values at the boundary of the bock as well as at the boundary of initial matrix 
in this case we take the id for(i,j) and put the same for id1,id2,id3,id4 accordingly.








*/
__global__ void step1(float* fn2, float* fn1, float B)
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
int threadId = blockId *(blockDim.x* blockDim.y)+(threadIdx.y *blockDim.x)+threadIdx.x;  
	
	//id of nearby points in case 1 :
	int id1 = threadId + 1 ;						
	int id2 = threadId - 1  ;
	
	
     	 int id3 =  threadId - blockDim.x ;  
     	 int id4 = threadId + blockDim.x ;
	
	//id of nearby points in case 2:
	if(threadIdx.y == 0) // here f(i,j) lies on the upper boundary of the block 
	{
	//block id of the block above the current one with f(i,j)	
	 int blockIdof3 = blockIdx.x + (blockIdx.y -1) * gridDim.x;
		
	//id of the point above the f(i,j) which lies on the block above it
	id3 = blockIdof3 *(blockDim.x* blockDim.y)+((blockDim.y-1) *blockDim.x)+threadIdx.x;  
	}
   	   

	if(threadIdx.y == blockDim.y -1 )  // here f(i,j) lies on the lower boundary of the block 
  	{
	//block id of the block above the current one with f(i,j)	
	int blockIdof4 = blockIdx.x + (blockIdx.y +1) * gridDim.x;
		
        //id of the point below the f(i,j) which lies on the block above it
	id4 = blockIdof4 *(blockDim.x* blockDim.y) + threadIdx.x;  
	}


	if(threadIdx.x == 0)  // here f(i,j) lies on the left boundary of the block
	{
	//block id of the block to the left the current one with f(i,j)
	 int blockIdof2 = (blockIdx.x-1) + blockIdx.y * gridDim.x;
		
	//id of the point to the left the f(i,j) which lies on the block beside it
	id2 = blockIdof2 *(blockDim.x* blockDim.y)+(threadIdx.y *blockDim.x)+(blockDim.x-1); 
	}
   	   

	if(threadIdx.x == blockDim.x -1 ) // here f(i,j) lies on the right boundary of the block
  	{
	//block id of the block to the left the current one with f(i,j)
	int blockIdof1 = (blockIdx.x+1) + blockIdx.y * gridDim.x;
	
	//id of the point to the left the f(i,j) which lies on the block beside it
	id1 = blockIdof1 *(blockDim.x* blockDim.y)+(threadIdx.y *blockDim.x);  
	}
	
	//3rd case where we take the id for f(i,j,t) and put it in id1,2,3,4 accordingly 
	if(threadIdx.y == 0 && blockIdx.y == 0)
   	  { id3 = threadId ; } 

	if(threadIdx.y == blockDim.y -1  && blockIdx.y == gridDim.y -1)
  	  { id4 = threadId ; } 

	if(threadIdx.x == 0 && blockIdx.x == 0)
   	  { id2 = threadId ; } 

	if(threadIdx.x == blockDim.x -1  && blockIdx.x == gridDim.x -1)
  	  { id1 = threadId ; } 


	

	
	
	
	// evaluating the first two terms of the equation and updating in f(i,j,t+1) i.e fn2
        fn2[threadId] = 2*fn1[threadId]+B*(fn1[id1]+ fn1[id2]+ fn1[id3] + fn1[id4] - (4*fn1[threadId]) );	

	
}





// step 2 involves subtraction of 3rd term in RHS in the equation 
__global__ void step2(float* fn2, float* fn0)
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
int threadId = blockId *(blockDim.x* blockDim.y)+(threadIdx.y *blockDim.x)+threadIdx.x;

	fn2[threadId] = fn2[threadId] - fn0[threadId] ;
}
__global__ void copy(float* fn1, float* fn0)
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
int threadId = blockId *(blockDim.x* blockDim.y)+(threadIdx.y *blockDim.x)+threadIdx.x;

	fn1[threadId] = fn0[threadId] ;
} 

					  



int main()
{	
	// intital matrix will be square matrix with dimensions N x N 
	int N = 1028;  // initalize the value of N	
	int t = 16;    // txt is the dimension length of block i.e txt is the number of threads 	
	int G = 64;	// GxG is the numnber of blocks in a grid 
	int size = N*N* sizeof(int);
	
	float f0[N*N], f1[N*N], f2[N*N],*fn1, *fn2,*fn0;  // initialize the three 1 D array 
	// here indexing goes according to bloks to block
	
	// intialize B 
	float h = 0.001  , At = 0.1 , c = 0.01  ,b ;
	b = (c*c*At*At) / (h*h) ;
	b= 0.0025; 
	


	//initialze the 1d array
	int i,j;
	
	for(j=0;j<N*N;j++){
			f0[j]=0;
			f1[j]=0;
			f2[j]=0;
				}
	int x = N*N/2;
	f1[x] = 5 ;   // this is the point where disturbance in intitalized






	hipMalloc(&fn1, size);
	hipMemcpy(fn1, f1 , size, hipMemcpyHostToDevice); 

	hipMalloc(&fn0, size);
	hipMemcpy(fn0, f0, size, hipMemcpyHostToDevice);

	hipMalloc(&fn2, size);

	     
	dim3   DimBlock(t,t);  	
	dim3   DimGrid(G,G);  
	
	int n;      // n is the number of iteration 
	n = 4 ;
	


	for(i=0 ;i<n ; i++)
	{
	
	step1<<< DimGrid,DimBlock >>>(fn2 , fn1 ,b);
	step2<<<DimGrid,DimBlock >>>(fn2 , fn1);
	
		
	copy<<< DimGrid,DimBlock >>>(fn0 , fn1 );
	copy<<< DimGrid,DimBlock >>>(fn1 , fn2 );
	
	}
		
		


		hipMemcpy(f2, fn2, size, hipMemcpyDeviceToHost);
		hipMemcpy(f1, fn1, size, hipMemcpyDeviceToHost);
		hipMemcpy(f0, fn0, size, hipMemcpyDeviceToHost);
  

for(i=0 ; i<N ; i++)
	{
  		 for(j=0;j<N;j++){
			  int index = ((t*t)* ((j/t) + (i/t)*G)) + j%t + ((i%t)*t);
			printf("%f  ",f2[index]);
				}
			 printf("\n");
	}
		
}
