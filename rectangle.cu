
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void step1(float* fn2, float* fn1, float B)
{
	int id = (threadIdx.y * blockDim.x) + threadIdx.x + (blockIdx.x * blockDim.x * blockDim.y); 
	int id1 = id + 1 ;						
	int id2 = id -1 ;
	
	
   int id3 =(threadIdx.y * blockDim.x) + threadIdx.x - blockDim.x + (blockIdx.x * blockDim.x * blockDim.y);  
   int id4 =(threadIdx.y * blockDim.x) + threadIdx.x + blockDim.x + (blockIdx.x * blockDim.x * blockDim.y) ;
	if(threadIdx.y == 0 && blockIdx.x ==0 )
     	{ id3 = id ; } 

	if(threadIdx.y == blockDim.y-1 && blockIdx.x == gridDim.x - 1)
     	{ id4 = id ; } 

		//printf("%d id k upar %d k neche %d bagal me %d %d  for %d\n ", id , id3 , id4 , id2,id1,blockIdx.x );

	
	
	
        fn2[id] = 2*fn1[id]+B*(fn1[id1]+ fn1[id2]+ fn1[id3] + fn1[id4] - (4*fn1[id]) );	

	
}






__global__ void step2(float* fn2, float* fn0)
{
	int id= threadIdx.y * blockDim.x + threadIdx.x + blockIdx.x * blockDim.x * blockDim.y  ;
	  

	fn2[id ] = fn2[id]- fn0[id] ;
}
__global__ void copy(float* fn1, float* fn0)
{
	int id = threadIdx.y * blockDim.x + threadIdx.x +  (blockIdx.x * blockDim.x *blockDim.y) ;

	fn1[id] = fn0[id] ;
} 

					  



int main()
{	
	int A = 32 , B = 16 ;
	
	int size = A*B* sizeof(int);
	
	float f0[A*B], f1[A*B], f2[A*B],*fn1, *fn2,*fn0;
	

	float h = 0.001  , At = 0.1 , c = 0.01  ,b ;
	b = (c*c*At*At) / (h*h) ;
	b= 0.25;


	int i,j;
	for(i=0; i<B ; i++ )
	{
		for(j=0; j<A ; j++ )
		{
			f0[j + (i*A)] = 0;
			f1[j + (i*A)] = 0;
			f2[j + (i*A)] = 0;
		}
	}
	int x = (A*B)/2;
	f1[x] = 5 ;


	hipMalloc(&fn1, size);
	hipMemcpy(fn1, f1 , size, hipMemcpyHostToDevice); 

	hipMalloc(&fn0, size);
	hipMemcpy(fn0, f0, size, hipMemcpyHostToDevice);

	hipMalloc(&fn2, size);

	     
	dim3   DimBlock(A , B); 	
	dim3   DimGrid(1, 1);
	
	
	int n =3;
	for(i=0 ;i<n ; i++)
	{
	step1<<<  DimGrid,DimBlock >>>(fn2 , fn1 ,b);
	step2<<<  DimGrid,DimBlock >>>(fn2 , fn1);
	
		
	copy<<<  DimGrid,DimBlock >>>(fn0 , fn1 );
	copy<<<  DimGrid,DimBlock >>>(fn1 , fn2 );

	}
		
		

		hipMemcpy(f2, fn2, size, hipMemcpyDeviceToHost);
		hipMemcpy(f1, fn1, size, hipMemcpyDeviceToHost);
		hipMemcpy(f0, fn0, size, hipMemcpyDeviceToHost);
    
	for(i=13; i<19 ; i++ )
	{
		for(j=13; j<19 ; j++ )
		{
		printf("%f   " ,f2[j + (i*B)]);
		}
		printf("\n ");
	


	}
	printf("%f  %f %f %f %f\n end " ,f2[x],f2[x-32],f2[x+32], f2[x+65], f2[x-65]);
	
	
	
}
